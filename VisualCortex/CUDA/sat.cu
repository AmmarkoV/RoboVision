
//CUDA Kernel..
__global__  void addUpRow(char * pixel,unsigned int * SAT)
       {
          // threadIdx.x is a built-in variable  provided by CUDA at runtime
          int row = threadIdx.x;

          pixel += row * 320;

          int pixels_remaining = 319;
          unsigned int  * next_SAT = SAT + 1;
          while (pixels_remaining!=0)
           {
             *next_SAT = *SAT + *pixel;
             ++pixel;
             ++next_SAT;
             ++SAT;
             --pixels_remaining;
           }
       }

__global__  void addUpColumn(unsigned int * SAT)
       {
          // threadIdx.x is a built-in variable  provided by CUDA at runtime
          int column = threadIdx.x;

          SAT += column;

          int pixels_remaining = 239;
          unsigned int  * next_SAT = SAT + 320;
          while (pixels_remaining!=0)
           {
             *next_SAT += *SAT ;
             next_SAT+=320;
             SAT+=320;
             --pixels_remaining;
           }
       }


#include <hip/hip_runtime.h>
#include  <stdio.h>

#define PPMREADBUFLEN 256

struct Image
{
  char * pixels;
  unsigned int size_x;
  unsigned int size_y;
  unsigned int depth;
  unsigned int image_size;
};

int ReadPPM(char * filename,struct Image * pic)
{
    FILE *pf=0;
    pf = fopen(filename,"rb");

    if (pf!=0 )
    {
        char buf[PPMREADBUFLEN], *t;
        unsigned int w=0, h=0, d=0;
        int r=0;

        t = fgets(buf, PPMREADBUFLEN, pf);
        if ( (t == 0) || ( strncmp(buf, "P6\n", 3) != 0 ) ) { fclose(pf); return 0; }
        do
        { /* Px formats can have # comments after first line */
           t = fgets(buf, PPMREADBUFLEN, pf);
           if ( t == 0 ) { fclose(pf); return 0; }
        } while ( strncmp(buf, "#", 1) == 0 );
        r = sscanf(buf, "%u %u", &w, &h);
        if ( r < 2 ) { fclose(pf); return 0; }
        // The program fails if the first byte of the image is equal to 32. because
        // the fscanf eats the space and the image is read with some bit less
        r = fscanf(pf, "%u\n", &d);
        if ( (r < 1) || ( d != 255 ) ) { fclose(pf); return 0; }

        if ( (w!=pic->size_x) || (h!=pic->size_y) )
           {
             fprintf(stderr,"Incorrect file size ( %s ) :P\n",filename);
             if ( w * h > pic->size_x * pic->size_y )
               {
                 fprintf(stderr,"File %s will lead to overflow stopping read..\n",filename);
                 fclose(pf);
                 return 0;
               }
           }

        if ( pic->pixels != 0 )
        {
            size_t rd = fread(pic->pixels,3, w*h, pf);
            fclose(pf);
            if ( rd < w*h )
            {
               return 0;
            }
            return 1;
        }
        fclose(pf);
    }
  return 0;
}

int WritePPM(char * filename,struct Image * pic)
{

    FILE *fd=0;
    fd = fopen(filename,"wb");

    if (fd!=0)
	{
     unsigned int n=0;

     fprintf(fd, "P6\n%d %d\n255\n", pic->size_x, pic->size_y);
     n = (unsigned int ) ( pic->size_x * pic->size_y ) ;

     fwrite(pic->pixels, 3, n, fd);

     fflush(fd);
     fclose(fd);

     return 1;
	}

  return 0;
}



int  main()
{
 fprintf(stderr,"CUDA Enabled Summed Area Table implementation..\n");
 struct Image input_img={0};
 input_img.size_x=320;
 input_img.size_y=240;
 ReadPPM("cudatest.ppm",&input_img);

 char * pixels;
 int pixels_size= input_img.size_x * input_img.size_y * 3 * sizeof(char);

 unsigned int * SAT;
 int SAT_size= input_img.size_x * input_img.size_y * 3 * sizeof(unsigned int);

 unsigned int SAT_Local[320*240];


 hipMalloc((void**)&pixels, pixels_size);
 hipMalloc((void**)& SAT, SAT_size);

 hipMemcpy(pixels, input_img.pixels , pixels_size,  hipMemcpyHostToDevice);
 //CUDA
 addUpRow<<<1,320>>>(pixels, SAT);
 addUpColumn<<<1,240>>>(SAT);

 hipMemcpy(SAT_Local, &SAT, SAT_size,  hipMemcpyDeviceToHost);

 hipFree(pixels);
 hipFree(SAT);
}
