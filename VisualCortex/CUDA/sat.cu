#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

 * main.c

int WIDTH = 1920;
int HEIGHT = 1080;
int THREADS_PER_BLOCK_1 = 256;
int THREADS_PER_BLOCK_2 = 128;
#define REPEAT_TIMES 177

char * device_inputArray;
unsigned int * device_outputArray;

__global__
void sumRow1(char *inputArray, unsigned int inputByteSize, unsigned int rowSize,
		unsigned int *outputArray)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int index = row * rowSize;

	char * rowPos = inputArray + index;

	if (rowPos + rowSize < inputArray + inputByteSize)
	{
		outputArray[index] = inputArray[index];

		int i;
		int idx;
		for (i = 1; i < rowSize; i++)
		{
			idx = index + i;
			outputArray[idx] = outputArray[idx - 1] + inputArray[idx];
		}
	}
}

__global__
void sumColumn1(unsigned int *outputArray, unsigned int outputByteSize,
		unsigned int rowSize, unsigned int columnSize)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int index = column;

	unsigned int * columnPos = outputArray + index;
	unsigned int * columnLimit = columnPos + (rowSize * (columnSize - 1));

	if (columnLimit < outputArray + outputByteSize)
	{
		int i;
		int idx;
		int offset;
		for (i = 1; i < columnSize; i++)
		{
			offset = i * rowSize;
			idx = index + offset;
			outputArray[idx] = outputArray[idx - offset] + outputArray[idx];
		}
	}
}

__global__
void sumRow2(char *inputArray, unsigned int inputByteSize, unsigned int rowSize,
		unsigned int columnSize, unsigned int *outputArray)
{
	//         BLOCKID        32         THREADID
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < columnSize)
	{
		char * rowPos = inputArray + row * rowSize;
		char * rowLimit = rowPos + rowSize;

		unsigned int * outPrev = outputArray + row * rowSize;
		unsigned int * out = outPrev + 1;

		*outPrev = *rowPos;
		++rowPos;

		while (rowPos < rowLimit)
		{
			*out = *outPrev + *rowPos;
			++out;
			++outPrev;
			++rowPos;
		}
	}
}

__global__
void sumColumn2(unsigned int *outputArray, unsigned int outputByteSize,
		unsigned int rowSize, unsigned int columnSize)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < rowSize)
	{
		unsigned int * columnPos = outputArray + column;
		unsigned int * columnLimit = columnPos + (rowSize * (columnSize - 1));

		//rowSizeCONST
		unsigned int * outPrev = columnPos;
		unsigned int * out = columnPos + rowSize;

		while (out < columnLimit)
		{
			*out += *outPrev;
			out += rowSize;
			outPrev += rowSize;
		}
	}
}

void Preconditions_checkMemoryAllocation(void * array)
{
	if (!array)
	{
		printf("Memory allocation was not successful\n");
		exit(1);
	}
}


void printOutput(unsigned int * output, unsigned int width, unsigned int height)
{
	printf("Output \n");
	int x, y;
	for (y = 0; y < 7; y++)
	{
		for (x = 0; x < width; x++)
		{
			printf("%u ", output[y * width + x]);
		}
		printf("\n");
	}
}

int InitializeCUDASAT(int height, int width, int threads1, int threads2)
{
	WIDTH = width;
	HEIGHT = height;
	THREADS_PER_BLOCK_1 = threads1;
	THREADS_PER_BLOCK_2 = threads2;

	unsigned int inputByteSize = WIDTH * HEIGHT * sizeof(char);
	hipMalloc((void**) &device_inputArray, inputByteSize);

	unsigned int outputByteSize = WIDTH * HEIGHT * sizeof(unsigned int);
	hipMalloc((void**) &device_outputArray, outputByteSize);

	return 1;
}

int closeCUDASAT()
{
	hipFree(device_inputArray);
	hipFree(device_outputArray);

	return 1;
}

int main()
{
	InitializeCUDASAT(WIDTH, HEIGHT, THREADS_PER_BLOCK_1, THREADS_PER_BLOCK_2);

	unsigned int inputByteSize = WIDTH * HEIGHT * sizeof(char);
	char * inputArray = (char *) malloc(inputByteSize);
	Preconditions_checkMemoryAllocation((void*) inputArray);

	unsigned int outputByteSize = WIDTH * HEIGHT * sizeof(unsigned int);
	unsigned int * outputArray = (unsigned int *) malloc(outputByteSize);
	Preconditions_checkMemoryAllocation((void*) outputArray);

	int i = 0;

	for (i = 0; i < REPEAT_TIMES; i++)
	{

		memset(inputArray, 1, inputByteSize);
		hipMemcpy(device_inputArray, inputArray, inputByteSize,
				hipMemcpyHostToDevice);

		int Blocks = (HEIGHT - 1) / THREADS_PER_BLOCK_1 + 1;
		printf("Gonna use %u blocks and %u threads for sumRow\n", Blocks,
				THREADS_PER_BLOCK_1);
		sumRow2<<<Blocks, THREADS_PER_BLOCK_1>>>(device_inputArray,
				inputByteSize, WIDTH, HEIGHT, device_outputArray);

		Blocks = (WIDTH - 1) / THREADS_PER_BLOCK_2 + 1;
		printf("Gonna use %u blocks and %u threads for sumRow\n", Blocks,
				THREADS_PER_BLOCK_2);
		sumColumn2<<<Blocks, THREADS_PER_BLOCK_2>>>(device_outputArray,
				outputByteSize, WIDTH, HEIGHT);

		hipMemcpy(outputArray, device_outputArray, outputByteSize,
				hipMemcpyDeviceToHost);

	}

	printOutput(outputArray, WIDTH, HEIGHT);

	closeCUDASAT();

	free(inputArray);
	free(outputArray);
}

