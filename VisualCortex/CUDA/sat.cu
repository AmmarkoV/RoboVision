
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>


int WIDTH = 1920;
int HEIGHT = 1080;
int THREADS_PER_BLOCK_1 = 256;
int THREADS_PER_BLOCK_2 = 128;
#define REPEAT_TIMES 1

char * device_inputArray;
unsigned int * device_outputArray;
__global__
void sumRow1(char *inputArray, unsigned int inputByteSize,
		unsigned int totalRows, unsigned int totalColumns,
		unsigned int *outputArray)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < totalRows)
	{
		int index = row * totalColumns;
		outputArray[index] = inputArray[index];

		int i;
		int idx;
		for (i = 1; i < totalColumns; i++)
		{
			idx = index + i;
			outputArray[idx] = outputArray[idx - 1] + inputArray[idx];
		}
	}
}

__global__
void sumColumn1(unsigned int *outputArray, unsigned int outputByteSize,
		unsigned int totalRows, unsigned int totalColumns)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < totalColumns)
	{
		int i;
		int offset;
		int prevOffset;
		for (i = 1; i < totalRows; i++)
		{
			prevOffset = (i - 1) * totalColumns;
			offset = prevOffset + totalColumns;
			outputArray[column + offset] = outputArray[column + prevOffset]
					+ outputArray[column + offset];
		}
	}
}
__global__
void sumRow2(char *inputArray, unsigned int inputByteSize,
		unsigned int totalRows, unsigned int totalColumns,
		unsigned int *outputArray)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < totalRows)
	{
		char * rowPos = inputArray + row * totalColumns;
		char * rowLimit = rowPos + totalColumns;

		unsigned int * outPrev = outputArray + row * totalColumns;
		unsigned int * out = outPrev + 1;

		*outPrev = *rowPos;
		++rowPos;

		while (rowPos < rowLimit)
		{
			*out = *outPrev + *rowPos;
			++out;
			++outPrev;
			++rowPos;
		}
	}
}

__global__
void sumColumn2(unsigned int *outputArray, unsigned int outputByteSize,
		unsigned int totalRows, unsigned int totalColumns)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	if (column < totalColumns)
	{
		unsigned int * columnPos = outputArray + column;
		unsigned int * columnLimit = columnPos
				+ (totalColumns * (totalRows - 1));

		unsigned int * outPrev = columnPos;
		unsigned int * out = columnPos + totalColumns;

		while (out <= columnLimit)
		{
			*out += *outPrev;
			out += totalColumns;
			outPrev += totalColumns;
		}
	}
}

void Preconditions_checkMemoryAllocation(void * array)
{
	if (!array)
	{
		printf("Memory allocation was not successful\n");
		exit(1);
	}
}

void printOutput(unsigned int * output, unsigned int width, unsigned int height)
{
	printf("Output \n");
	int x, y;
	for (y = 0; y < height; y++)
	{
		for (x = 0; x < width; x++)
		{
			printf("%u ", output[y * width + x]);
		}
		printf("\n");
	}
}

int InitializeCUDASAT(int height, int width, int threads1, int threads2)
{
	WIDTH = width;
	HEIGHT = height;
	THREADS_PER_BLOCK_1 = threads1;
	THREADS_PER_BLOCK_2 = threads2;

	unsigned int inputByteSize = WIDTH * HEIGHT * sizeof(char);
	hipMalloc((void**) &device_inputArray, inputByteSize);

	unsigned int outputByteSize = WIDTH * HEIGHT * sizeof(unsigned int);
	hipMalloc((void**) &device_outputArray, outputByteSize);

	return 1;
}

int closeCUDASAT()
{
	hipFree(device_inputArray);
	hipFree(device_outputArray);

	return 1;
}

int main()
{
	InitializeCUDASAT(WIDTH, HEIGHT, THREADS_PER_BLOCK_1, THREADS_PER_BLOCK_2);

	unsigned int inputByteSize = WIDTH * HEIGHT * sizeof(char);
	char * inputArray = (char *) malloc(inputByteSize);
	Preconditions_checkMemoryAllocation((void*) inputArray);

	unsigned int outputByteSize = WIDTH * HEIGHT * sizeof(unsigned int);
	unsigned int * outputArray = (unsigned int *) malloc(outputByteSize);
	Preconditions_checkMemoryAllocation((void*) outputArray);

	int i = 0;

	for (i = 0; i < REPEAT_TIMES; i++)
	{

		memset(inputArray, 1, inputByteSize);
		hipMemcpy(device_inputArray, inputArray, inputByteSize,
				hipMemcpyHostToDevice);

		int Blocks = (HEIGHT - 1) / THREADS_PER_BLOCK_1 + 1;
		printf("Gonna use %u blocks and %u threads for sumRow\n", Blocks,
				THREADS_PER_BLOCK_1);
		sumRow1<<<(HEIGHT - 1) / THREADS_PER_BLOCK_1 + 1, THREADS_PER_BLOCK_1>>>( device_inputArray, inputByteSize, HEIGHT, WIDTH, device_outputArray);

		Blocks = (WIDTH - 1) / THREADS_PER_BLOCK_2 + 1;
		printf("Gonna use %u blocks and %u threads for sumRow\n", Blocks,
				THREADS_PER_BLOCK_2);
		sumColumn1<<<(WIDTH - 1) / THREADS_PER_BLOCK_2 + 1, THREADS_PER_BLOCK_2>>>( device_outputArray, outputByteSize, HEIGHT, WIDTH);





		hipMemcpy(outputArray, device_outputArray, outputByteSize,
				hipMemcpyDeviceToHost);

	}

	//printOutput(outputArray, WIDTH, HEIGHT);

	closeCUDASAT();

	free(inputArray);
	free(outputArray);
}

