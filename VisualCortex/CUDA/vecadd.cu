   //  Kernel definition, see also section 4.2.3 of Nvidia Cuda Programming Guide
       __global__  void vecAdd(float* A, float* B, float* C)
       {
          // threadIdx.x is a built-in variable  provided by CUDA at runtime
          int i = threadIdx.x;
          A[i]=0;
          B[i]=i;
          C[i] = A[i] + B[i];
       }




#include <hip/hip_runtime.h>
#include  <stdio.h>
#define  SIZE 100

#define PPMREADBUFLEN 256

struct Image
{
  char * pixels;
  unsigned int size_x;
  unsigned int size_y;
  unsigned int depth;
  unsigned int image_size;
};

int ReadPPM(char * filename,struct Image * pic)
{
    FILE *pf=0;
    pf = fopen(filename,"rb");

    if (pf!=0 )
    {
        char buf[PPMREADBUFLEN], *t;
        unsigned int w=0, h=0, d=0;
        int r=0;

        t = fgets(buf, PPMREADBUFLEN, pf);
        if ( (t == 0) || ( strncmp(buf, "P6\n", 3) != 0 ) ) { fclose(pf); return 0; }
        do
        { /* Px formats can have # comments after first line */
           t = fgets(buf, PPMREADBUFLEN, pf);
           if ( t == 0 ) { fclose(pf); return 0; }
        } while ( strncmp(buf, "#", 1) == 0 );
        r = sscanf(buf, "%u %u", &w, &h);
        if ( r < 2 ) { fclose(pf); return 0; }
        // The program fails if the first byte of the image is equal to 32. because
        // the fscanf eats the space and the image is read with some bit less
        r = fscanf(pf, "%u\n", &d);
        if ( (r < 1) || ( d != 255 ) ) { fclose(pf); return 0; }

        if ( (w!=pic->size_x) || (h!=pic->size_y) )
           {
             fprintf(stderr,"Incorrect file size ( %s ) :P\n",filename);
             if ( w * h > pic->size_x * pic->size_y )
               {
                 fprintf(stderr,"File %s will lead to overflow stopping read..\n",filename);
                 fclose(pf);
                 return 0;
               }
           }

        if ( pic->pixels != 0 )
        {
            size_t rd = fread(pic->pixels,3, w*h, pf);
            fclose(pf);
            if ( rd < w*h )
            {
               return 0;
            }
            return 1;
        }
        fclose(pf);
    }
  return 0;
}

int WritePPM(char * filename,struct Image * pic)
{

    FILE *fd=0;
    fd = fopen(filename,"wb");

    if (fd!=0)
	{
     unsigned int n=0;

     fprintf(fd, "P6\n%d %d\n255\n", pic->size_x, pic->size_y);
     n = (unsigned int ) ( pic->size_x * pic->size_y ) ;

     fwrite(pic->pixels, 3, n, fd);

     fflush(fd);
     fclose(fd);

     return 1;
	}

  return 0;
}



int  main()
{
 struct Image input_img={0};
 input_img.size_x=320;
 input_img.size_y=240;
 ReadPPM("cudatest.ppm",&input_img);

 int N=SIZE;
 float A[SIZE], B[SIZE], C[SIZE];
 char * pixels;
 int pixels_size= input_img.size_x * input_img.size_y * 3 * sizeof(char);

 float *devPtrA; float *devPtrB; float *devPtrC;
 int memsize= SIZE * sizeof(float);


 hipMalloc((void**)&pixels, memsize);
 hipMalloc((void**)&devPtrA, memsize);
 hipMalloc((void**)&devPtrB, memsize);
 hipMalloc((void**)&devPtrC, memsize);
 hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice);
 hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice);
 // __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
 vecAdd<<<1, N>>>(devPtrA,  devPtrB, devPtrC);
 hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost);

  for (int i=0; i<SIZE; i++)
            printf("C[%d]=%f\n",i,C[i]);

            hipFree(devPtrA);
           hipFree(devPtrA);
           hipFree(devPtrA);
}
